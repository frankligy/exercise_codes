#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"
//#include "utils.h"

const int N= 1024;	// matrix size will be NxN
#define TILE_DIM    16
#define BLOCK_ROWS  16

int compare_matrices(float *gpu, float *ref, int N)
{
        int result = 0;
        for(int j=0; j < N; j++)
        for(int i=0; i < N; i++)
                if (ref[i + j*N] != gpu[i + j*N])
                   {result = 1;}
 return result;
}


// fill a matrix with sequential numbers in the range 0..N-1
void fill_matrix(float *mat, int N)
{
        for(int j=0; j < N * N; j++)
                mat[j] = (float) j;
}

// The following functions and kernels are for your references
void 
transpose_CPU(float in[], float out[])
{
	for(int j=0; j < N; j++)
    	for(int i=0; i < N; i++)
      		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched on a single thread
__global__ void 
transpose_serial(float in[], float out[])
{
	for(int j=0; j < N; j++)
		for(int i=0; i < N; i++)
			out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per row of output matrix
__global__ void 
transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	for(int j=0; j < N; j++)
		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}



// Write two tiled versions of transpose -- One using shared memory. 
// To be launched with one thread per element, in KxK threadblocks.
// You will determine for each thread (x,y) in tile the element (i,j) of global output matrix. 

__global__ void 
transpose_parallel_per_element_tiled(float in[], float out[])
{
	int x = blockIdx.x * 16 + threadIdx.x;
	int y = blockIdx.y * 16 + threadIdx.y;
	int width = gridDim.x * 16;
  
	for (int j = 0; j < 16; j+= 16)
	  out[x*width + (y+j)] = in[(y+j)*width + x];
}

__global__ void 
transpose_parallel_per_element_tiled_shared(float in[], float out[])
{
    __shared__ float tile[TILE_DIM][TILE_DIM];
  
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
  
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
       tile[threadIdx.y+j][threadIdx.x] = in[(y+j)*width + x];
  
    __syncthreads();
  
    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;
  
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
       out[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
  }

/*
let's say you process ./program.cpp a b c
argc = 4   (1 + number of argument)
argv = [a,b,c]
*/


int main(int argc, char **argv)
{
	int numbytes = N * N * sizeof(float);

	float *in = (float *) malloc(numbytes);
	float *out = (float *) malloc(numbytes);
	float *gold = (float *) malloc(numbytes);

	fill_matrix(in, N);
	transpose_CPU(in, gold);

	float *d_in, *d_out;

	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

	GpuTimer timer;


    timer.Start();
	transpose_serial<<<1,1>>>(d_in, d_out);
	timer.Stop();
    for (int i=0; i < N*N; ++i){out[i] = 0.0;}
    hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_serial: %g ms.\nVerifying ...%s\n", 
		   timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

   
    hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
    timer.Start();
	transpose_parallel_per_row<<<1,N>>>(d_in, d_out);
	timer.Stop();
    for (int i=0; i < N*N; ++i){out[i] = 0.0;}  //clean out
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_row: %g ms.\nVerifying ...%s\n", 
		    timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

    hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
    // Tiled versions
    const int K= 16;
    dim3 blocks_tiled(N/K,N/K);
	dim3 threads_tiled(K,K);
	timer.Start();
	transpose_parallel_per_element_tiled<<<blocks_tiled,threads_tiled>>>(d_in, d_out);
	timer.Stop();
    for (int i=0; i < N*N; ++i){out[i] = 0.0;}
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled %dx%d: %g ms.\nVerifying ...%s\n", 
		   K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

    hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
    dim3 blocks_tiled_sh(N/K,N/K);
	dim3 threads_tiled_sh(K,K);
    timer.Start();
	transpose_parallel_per_element_tiled_shared<<<blocks_tiled_sh,threads_tiled_sh>>>(d_in, d_out);
	timer.Stop();
    for (int i=0; i < N*N; ++i){out[i] = 0.0;}
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled_shared %dx%d: %g ms.\nVerifying ...%s\n", 
		   K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

	hipFree(d_in);
	hipFree(d_out);
}
